#include "hip/hip_runtime.h"
/**
 * Mandelbrot 分形渲染器 - CUDA GPU实现
 * 
 * 本文件实现了基于CUDA的GPU并行渲染
 * GPU优化策略:
 * 1. 每个线程计算一个像素
 * 2. 线程块大小优化 (16x16 = 256线程)
 * 3. 内存合并访问模式
 * 4. 避免分支发散
 * 5. 寄存器使用优化
 * 
 * 硬件目标: RTX 3090 (8704 CUDA Cores, 24GB VRAM)
 * 
 * 作者: Geoffrey Wang (with Claude AI assistance)
 * 日期: 2025-08-12
 */

#include "../include/render_cuda.hpp"
#include <iostream>
#include <chrono>
#include <sstream>
#include <algorithm>
#include <vector>
#include <stdexcept>

namespace MandelbrotCUDA {

    /**
     * CUDA Device函数: 优化的Mandelbrot迭代计算
     * 运行在GPU上，每个线程调用一次
     */
    __device__ int mandelbrot_iterations_cuda(double cx, double cy, int max_iter) {
        double zx = 0.0, zy = 0.0;
        double zx2, zy2;
        int iter = 0;
        
        // 优化的迭代循环，减少浮点运算
        while (iter < max_iter) {
            zx2 = zx * zx;
            zy2 = zy * zy;
            
            // 早期退出条件: |z|^2 > 4
            if (zx2 + zy2 > 4.0) break;
            
            // z = z^2 + c
            zy = 2.0 * zx * zy + cy;
            zx = zx2 - zy2 + cx;
            ++iter;
        }
        
        return iter;
    }

    /**
     * CUDA Device函数: 迭代次数转RGB颜色
     * 使用快速HSV到RGB转换
     */
    __device__ void iterations_to_rgb_cuda(int iterations, int max_iter, 
                                          unsigned char* r, unsigned char* g, unsigned char* b) {
        if (iterations == max_iter) {
            // 属于Mandelbrot集合内部
            *r = *g = *b = 0;
            return;
        }
        
        // 归一化迭代值
        double t = (double)iterations / max_iter;
        
        // 彩虹色谱映射 (优化版本)
        int color_r, color_g, color_b;
        
        if (t < 0.16) {
            color_r = 255;
            color_g = (int)(255 * t / 0.16);
            color_b = 0;
        } else if (t < 0.33) {
            color_r = (int)(255 * (0.33 - t) / 0.17);
            color_g = 255;
            color_b = 0;
        } else if (t < 0.5) {
            color_r = 0;
            color_g = 255;
            color_b = (int)(255 * (t - 0.33) / 0.17);
        } else if (t < 0.66) {
            color_r = 0;
            color_g = (int)(255 * (0.66 - t) / 0.16);
            color_b = 255;
        } else if (t < 0.83) {
            color_r = (int)(255 * (t - 0.66) / 0.17);
            color_g = 0;
            color_b = 255;
        } else {
            color_r = 255;
            color_g = 0;
            color_b = (int)(255 * (1.0 - t) / 0.17);
        }
        
        *r = (unsigned char)color_r;
        *g = (unsigned char)color_g;
        *b = (unsigned char)color_b;
    }

    /**
     * CUDA Kernel: 并行Mandelbrot渲染
     * 每个线程处理一个像素
     */
    __global__ void mandelbrot_kernel(unsigned char* output,
                                     int width, int height,
                                     double x_min, double x_max,
                                     double y_min, double y_max,
                                     int max_iter) {
        // 计算当前线程对应的像素坐标
        int px = blockIdx.x * blockDim.x + threadIdx.x;
        int py = blockIdx.y * blockDim.y + threadIdx.y;
        
        // 边界检查
        if (px >= width || py >= height) return;
        
        // 像素坐标到复平面坐标的映射
        double cx = x_min + (x_max - x_min) * px / (width - 1);
        double cy = y_min + (y_max - y_min) * py / (height - 1);
        
        // 计算迭代次数
        int iterations = mandelbrot_iterations_cuda(cx, cy, max_iter);
        
        // 转换为RGB颜色
        unsigned char r, g, b;
        iterations_to_rgb_cuda(iterations, max_iter, &r, &g, &b);
        
        // 存储结果 (行主序存储，RGB交错)
        int pixel_index = (py * width + px) * 3;
        output[pixel_index] = r;     // Red
        output[pixel_index + 1] = g; // Green
        output[pixel_index + 2] = b; // Blue
    }

    // C接口包装函数
    extern "C" void mandelbrot_kernel_launcher(unsigned char* output,
                                              int width, int height,
                                              double x_min, double x_max,
                                              double y_min, double y_max,
                                              int max_iter,
                                              dim3 grid_size, dim3 block_size) {
        mandelbrot_kernel<<<grid_size, block_size>>>(output, width, height,
                                                     x_min, x_max, y_min, y_max, max_iter);
    }

    std::vector<GPUInfo> get_available_gpus() {
        std::vector<GPUInfo> gpus;
        int device_count;
        
        CUDA_CHECK(hipGetDeviceCount(&device_count));
        
        for (int i = 0; i < device_count; ++i) {
            hipDeviceProp_t prop;
            CUDA_CHECK(hipGetDeviceProperties(&prop, i));
            
            // 获取内存信息
            size_t free_mem, total_mem;
            hipSetDevice(i);
            CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));
            
            GPUInfo info;
            info.device_id = i;
            info.name = prop.name;
            info.total_memory = total_mem / (1024 * 1024);  // MB
            info.free_memory = free_mem / (1024 * 1024);    // MB
            info.compute_capability_major = prop.major;
            info.compute_capability_minor = prop.minor;
            info.multiprocessor_count = prop.multiProcessorCount;
            info.max_threads_per_block = prop.maxThreadsPerBlock;
            info.max_threads_per_multiprocessor = prop.maxThreadsPerMultiProcessor;
            info.warp_size = prop.warpSize;
            
            gpus.push_back(info);
        }
        
        return gpus;
    }

    int get_best_gpu_device() {
        auto gpus = get_available_gpus();
        if (gpus.empty()) {
            throw std::runtime_error("没有找到可用的GPU设备");
        }
        
        // 选择计算能力最高的GPU
        int best_device = 0;
        int best_score = 0;
        
        for (const auto& gpu : gpus) {
            int score = gpu.compute_capability_major * 100 + gpu.compute_capability_minor * 10 + 
                       (gpu.multiprocessor_count / 10);
            if (score > best_score) {
                best_score = score;
                best_device = gpu.device_id;
            }
        }
        
        return best_device;
    }

    void initialize_cuda(int device_id) {
        int device_count;
        CUDA_CHECK(hipGetDeviceCount(&device_count));
        
        if (device_id >= device_count) {
            throw std::runtime_error("无效的GPU设备ID: " + std::to_string(device_id));
        }
        
        CUDA_CHECK(hipSetDevice(device_id));
        
        // 预热GPU
        CUDA_CHECK(hipFree(0));
        
        std::cout << "[CUDA] 初始化设备 " << device_id << " 完成" << std::endl;
    }

    void cleanup_cuda() {
        CUDA_CHECK(hipDeviceReset());
    }

    std::pair<size_t, size_t> get_gpu_memory_info(int device_id) {
        hipSetDevice(device_id);
        
        size_t free_mem, total_mem;
        CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));
        
        return {(total_mem - free_mem) / (1024 * 1024), total_mem / (1024 * 1024)};
    }

    std::pair<dim3, dim3> calculate_optimal_block_config(int width, int height) {
        // 根据图像尺寸选择最优的线程块大小
        dim3 block_size;
        
        if (width >= 1024 && height >= 1024) {
            block_size = dim3(32, 32);  // 1024线程/块，适用于大图像
        } else if (width >= 512 && height >= 512) {
            block_size = dim3(16, 16);  // 256线程/块，适用于中等图像
        } else {
            block_size = dim3(8, 8);    // 64线程/块，适用于小图像
        }
        
        // 计算网格大小
        dim3 grid_size((width + block_size.x - 1) / block_size.x,
                       (height + block_size.y - 1) / block_size.y);
        
        return {block_size, grid_size};
    }

    std::string get_cuda_info() {
        std::ostringstream info;
        
        int runtime_version, driver_version;
        CUDA_CHECK(hipRuntimeGetVersion(&runtime_version));
        CUDA_CHECK(hipDriverGetVersion(&driver_version));
        
        info << "CUDA Runtime版本: " << runtime_version << std::endl;
        info << "CUDA Driver版本: " << driver_version << std::endl;
        
        auto gpus = get_available_gpus();
        info << "可用GPU数量: " << gpus.size() << std::endl;
        
        for (const auto& gpu : gpus) {
            info << "设备 " << gpu.device_id << ": " << gpu.name << std::endl;
            info << "  计算能力: " << gpu.compute_capability_major << "." << gpu.compute_capability_minor << std::endl;
            info << "  多处理器: " << gpu.multiprocessor_count << " 个" << std::endl;
            info << "  总内存: " << gpu.total_memory << " MB" << std::endl;
            info << "  可用内存: " << gpu.free_memory << " MB" << std::endl;
        }
        
        return info.str();
    }

    std::vector<unsigned char> render_mandelbrot_cuda(const RenderParams& params, 
                                                      int device_id, 
                                                      int block_size) {
        std::cout << "[CUDA] 开始GPU渲染 Mandelbrot 集合..." << std::endl;
        std::cout << "[CUDA] 分辨率: " << params.width << "x" << params.height << std::endl;
        std::cout << "[CUDA] 最大迭代: " << params.max_iter << std::endl;
        std::cout << "[CUDA] 复平面范围: [" << params.x_min << "," << params.x_max 
                  << "] x [" << params.y_min << "," << params.y_max << "]" << std::endl;
        
        // 初始化CUDA
        initialize_cuda(device_id);
        
        // 显示GPU信息
        auto gpus = get_available_gpus();
        if (device_id < gpus.size()) {
            const auto& gpu = gpus[device_id];
            std::cout << "[CUDA] 使用GPU: " << gpu.name 
                      << " (计算能力 " << gpu.compute_capability_major << "." << gpu.compute_capability_minor << ")"
                      << std::endl;
        }
        
        auto start_time = std::chrono::high_resolution_clock::now();
        
        // 分配主机内存
        int total_pixels = params.width * params.height;
        size_t image_size = total_pixels * 3 * sizeof(unsigned char);
        std::vector<unsigned char> host_image(total_pixels * 3);
        
        // 分配GPU内存
        unsigned char* device_image;
        CUDA_CHECK(hipMalloc(&device_image, image_size));
        
        // 计算最优线程配置
        auto [block_dim, grid_dim] = calculate_optimal_block_config(params.width, params.height);
        
        std::cout << "[CUDA] 线程配置: " << block_dim.x << "x" << block_dim.y << " 块大小, "
                  << grid_dim.x << "x" << grid_dim.y << " 网格大小" << std::endl;
        std::cout << "[CUDA] 总线程数: " << block_dim.x * block_dim.y * grid_dim.x * grid_dim.y << std::endl;
        
        // 启动CUDA kernel
        mandelbrot_kernel<<<grid_dim, block_dim>>>(device_image,
                                                   params.width, params.height,
                                                   params.x_min, params.x_max,
                                                   params.y_min, params.y_max,
                                                   params.max_iter);
        
        // 检查kernel执行是否成功
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        
        // 将结果从GPU复制到CPU
        CUDA_CHECK(hipMemcpy(host_image.data(), device_image, image_size, hipMemcpyDeviceToHost));
        
        // 清理GPU内存
        CUDA_CHECK(hipFree(device_image));
        
        auto end_time = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
        
        std::cout << "[CUDA] 渲染完成! 耗时: " << duration.count() / 1000.0 << " ms" << std::endl;
        std::cout << "[CUDA] 性能: " << (total_pixels * 1000000.0 / duration.count()) << " 像素/秒" << std::endl;
        
        // 显示内存使用情况
        auto [used_mem, total_mem] = get_gpu_memory_info(device_id);
        std::cout << "[CUDA] GPU内存使用: " << used_mem << " / " << total_mem << " MB" << std::endl;
        
        return host_image;
    }

} // namespace MandelbrotCUDA
